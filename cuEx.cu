#include "hip/hip_runtime.h"
/*
 * Author: Quynh Ngo
 * @Copyright: Quynh Ngo, except the two functions HandleError, define HANDLE_ERROR, and the random function in Cuda
 * @Cuda version to run forest fire model on a network
 */

#ifndef CUDA_H
#define CUDA_H

#include <stdio.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "directedgraph.h"

/* These function are not mine, they belong to some books, some sites I read */
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ )) 


/* this GPU kernel function is used to initialize the random states */
// I modified from blockIdx.x to be the index of the thread
__global__ void init(unsigned int seed, hiprandState_t* states) 
{
	int Index =  blockIdx.x * blockDim.x + threadIdx.x;
	/* we have to initialize the state */
	hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
				 Index, //blockIdx.x, /* the sequence number should be different for each core (unless you want all
								//cores to get the same sequence of numbers for some reason - use thread id! */
				0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
				&states[Index]); // blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, unsigned int* numbers) 
{
	/* hiprand works like rand - except that it takes a state as a parameter */
	numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % 100;
}

/* CUDA's random number library uses hiprandState_t to keep track of the seed value
we will store a random state for every thread  */
// hiprandState_t* states;

/* allocate space on the GPU for the random states */
// hipMalloc((void**) &states, N * sizeof(hiprandState_t));

/* invoke the GPU to initialize all of the random states */
// init<<<N, 1>>>(time(0), states);

/* allocate an array of unsigned ints on the CPU and GPU */
// unsigned int cpu_nums[N];
// unsigned int* gpu_nums;
// hipMalloc((void**) &gpu_nums, N * sizeof(unsigned int));

/* invoke the kernel to get some random numbers */
// randoms<<<N, 1>>>(states, gpu_nums);

/* copy the random numbers back */
// hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
/*

/*******************************************************************************************/


/**** From now on, it's all mine ***********************************************************/

#define NumbCoactivationMatrices 6400

// think of kernel that output an array with the work of different thread
// with that the time series will be a big one that goes beyond the scope of a heap, too big 6553600 Gb.

// SO I HAVE TO DIVIDE THE GRID INTO SUB-GRIDS with the global function in CPU and have a loop to call it;
// It's doable;
// Each grid point is a thread
// that mean this thread should output the coactivation array portion


// Cuda version
// I need to collect list of co-activation matrices in form of an array
// Each thread will be in charge of a chunk
// The graph will be passed to global memory in GPU
// Each thread will handle one parameter setting for f, p
// The co-activation sub-array will be updated on the fly along side with


// This is the cuda version of forest fire model 
// param[in] graph which includes NumbNode, LinkListSize, LinkList from global memory
// param[in] f, p are the parameter values
// param[in] NumberOfTimeStep is from user-defined
// param[out] CoactivationMatrix

/*
 * @des return the index in the co-activation array which is actually only the upper right part of the matrix
 * @param[in] KernelIndex is the index of the kernel that are excecuting the function
 * @param[in] RowIndex is the row index of the element
 * @param[in] ColumnIndex is the column index of the element
 * @param[in] SizeOfMatrix is the size of the co-activation matrix
 * @note sizeMat*(sizeMat-1)/2 is the element of the upper part of the matrix;
 */	
__device__ int returnIndex(int KernelIndex, int RowIndex, int ColumnIndex, int SizeOfMatrix)
			{
				if(RowIndex == 0)
				{
					return  KernelIndex * SizeOfMatrix * (SizeOfMatrix-1)/2 + ColumnIndex - 1;
				}
				else
				{
					int Sum = 0;
					for(int i = 0; i < RowIndex - 1; i ++)
					{
						Sum += SizeOfMatrix - i - 1;
					}
					return KernelIndex * SizeOfMatrix * (SizeOfMatrix-1)/2 + Sum + ColumnIndex - RowIndex - 1;
				}
			}
	
/* Now the code for the kernel */

__global__ void calCoactivationMat(int *MaxTimeStep, 
									const int *NumbNode, const int *LinkListSize, 
									const int *LinkList,  const int *Dev_FirstLocationNeighbor, 
									int *Dev_PrevNodeState, int *Dev_CurrNodeState, hiprandState_t *states, float *CoactivationMatrix)
{

int Index =  blockIdx.x * blockDim.x + threadIdx.x;

while(Index < NumbCoactivationMatrices)
{
	double SpontaneousFiring = (double)threadIdx.x/(double)(double)blockDim.x;
	double RecoveryProbability = (double)blockIdx.x/(double)gridDim.x; 
	for(int j = 1;j < *MaxTimeStep; j++)
	{
		for(int i = 0; i< *NumbNode; i++)
		{
			float _CurrentStage = Dev_PrevNodeState[i + Index *(*NumbNode)]; 
			if(_CurrentStage == 1)
			{
				// next_time_step's_attribue;
				Dev_CurrNodeState[i + Index *(*NumbNode)] = 0;
			}
			else if(_CurrentStage == 0)
			{
				double Probability =  (double)(hiprand(&states[Index]) % 100)/100.0;
				if(Probability < RecoveryProbability)
				{ //recovery probability is p;
					Dev_CurrNodeState[i + Index *(*NumbNode)] = -1; 
				}
				else Dev_CurrNodeState[i + Index*(*NumbNode)] = 0; 
			}
			else
			{
				int LenghtNeighborNode1 = (int) LinkListSize[i];  // the lenght of adjacient list
										//of node1 (the number of adjacient nodes of node1)
				bool Excited = false;
				for(int k = 0; k< LenghtNeighborNode1; k++)
				{
					// now try to access the neighbor node
					int NextNodeName = LinkList[Dev_FirstLocationNeighbor[i] + k];
					if(Dev_PrevNodeState[NextNodeName] == 1)    
					{	
						Dev_CurrNodeState[i + Index*(*NumbNode)] = 1; 
						Excited = true;
						break;
					}
				}
				if(!Excited)
				{
					double Probability = (double)(hiprand(&states[Index]) % 100)/100.0; 
					if(Probability < SpontaneousFiring)
					{ 
						Dev_CurrNodeState[i + Index*(*NumbNode)] = 1;       
					}
					else Dev_CurrNodeState[i + Index*(*NumbNode)] = -1;     // Else update current stage to
					//be the next stage;
				}
			}// end else
		} // End second for loop

		// Update co-activation sum
		if(j > 1000)
		{ // discard the first 1000 time steps;
			for(int i = 0; i < *NumbNode; i++)
			{

				for( int k= i + 1; k < *NumbNode; k++ )
				{
					if( ( Dev_CurrNodeState[k + Index*(*NumbNode)] == Dev_CurrNodeState[i + Index*(*NumbNode)]) && (Dev_CurrNodeState[i + Index*(*NumbNode)] == 1))
					{
						CoactivationMatrix[returnIndex(Index, i, k, *NumbNode)] += 1;
					}	
				}
			}
		} // end if j > 100 

		for(int i = 0; i < *NumbNode; i ++)
		{
			Dev_PrevNodeState[i + Index*(*NumbNode)] = Dev_CurrNodeState[i + Index*(*NumbNode)];
		}
		
	} // End first for loop for max time step

	// Finish calculating co-activation
	for (int i = 0; i < *NumbNode; i ++)
	{
		for(int k = i + 1; k < *NumbNode; k ++)
		{
			CoactivationMatrix[returnIndex(Index, i, k, *NumbNode)] = CoactivationMatrix[returnIndex(Index, i, k, *NumbNode)]/(float)( *MaxTimeStep-1000); 
		}
	}
	//Go to increment 
  Index += blockDim.x * gridDim.x; // One thread can calculate many co-activation;	
 }// End while
}// End kernel

// Each thread handle their coactivation portion
	
/* The program */
	
extern "C" void extractCoactivationMatrices(const directedgraph &GraphAttach)
{
	
	/* allocate memory for graph in GPU */

	// pass number of node to device
	int *Host_NodeNumb = new int;
	*Host_NodeNumb = GraphAttach.getNumberNode();
	int *Dev_NodeNumb;
	HANDLE_ERROR(hipMalloc(&Dev_NodeNumb, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(Dev_NodeNumb, Host_NodeNumb, sizeof(int), hipMemcpyHostToDevice));

	// pass number of max time step to device
	int *Host_MaxTimeStep = new int;
	*Host_MaxTimeStep = 100000;
	int *Dev_MaxTimeStep;
	HANDLE_ERROR(hipMalloc(&Dev_MaxTimeStep, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(Dev_MaxTimeStep, Host_MaxTimeStep, sizeof(int), hipMemcpyHostToDevice));

	// pass array including number of neighbor nodes for each node to device
	int *Host_LinkListSize = new int[*Host_NodeNumb];
	std::vector<std::vector<int>> LinkList = GraphAttach.getInList();
	for(int i = 0; i < *Host_NodeNumb; i ++)
	{
		Host_LinkListSize[i] = (int)(LinkList[i]).size();
	}
	int *Dev_LinkListSize;
	HANDLE_ERROR(hipMalloc(&Dev_LinkListSize, (*Host_NodeNumb)*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(Dev_LinkListSize, Host_LinkListSize, (*Host_NodeNumb)*sizeof(int), hipMemcpyHostToDevice));

		
	// pass linklist to device
	int ListSize = 0;
	for(int i = 0; i < *Host_NodeNumb; i ++)
	{
		ListSize += Host_LinkListSize[i];
	}

	int Index = 0;
	int *Host_LinkList = new int[ListSize];
	for(int i = 0; i < *Host_NodeNumb; i ++)
	{ 
		for(int j = 0; j < (int)LinkList[i].size(); j++)
		{
			Host_LinkList[Index] = LinkList[i][j];
			Index += 1;
		}
	}
	int *Dev_LinkList;
	HANDLE_ERROR(hipMalloc(&Dev_LinkList, ListSize*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(Dev_LinkList, Host_LinkList, ListSize*sizeof(int), hipMemcpyHostToDevice));


	int *Host_FirstLocationNeighbor = new int[*Host_NodeNumb];
	for(int i = 0; i < *Host_NodeNumb; i ++)
	{
		int FirstLocationOfNodeI = 0;
		for(int j = 0; j < i; j ++)
		{
			FirstLocationOfNodeI += Host_LinkListSize[j];
		}
		Host_FirstLocationNeighbor[i] = FirstLocationOfNodeI;	
	}
	int *Dev_FirstLocationNeighbor;
	HANDLE_ERROR(hipMalloc(&Dev_FirstLocationNeighbor, (*Host_NodeNumb)*sizeof(int)));
	HANDLE_ERROR(hipMemcpy(Dev_FirstLocationNeighbor, Host_FirstLocationNeighbor, (*Host_NodeNumb)*sizeof(int), hipMemcpyHostToDevice));

	
	hiprandState_t* states;
	/* allocate space on the GPU for the random states */
	hipMalloc((void**) &states, NumbCoactivationMatrices * sizeof(hiprandState_t));

	/* invoke the GPU to initialize all of the random states */
	init<<<128, 128>>>(time(0), states);

	// Allocate the previous step time series, and the current step time series from the host
	// for memory in the device 
	// Done initializing stage of all nodes
	// Now then, update each time_step just by update_attribute;
	// For each node, check adjacient_list of this node,
	//considering the last entry of attribute to decide next stage of this node;
	// Each thread will be in charge of one co-activation or more :)
    

	
	srand(static_cast<unsigned int>(time(0))); //To create random numbers.
	int *Host_PrevNodeState = new int[(*Host_NodeNumb)*NumbCoactivationMatrices];

	for(int k = 0; k < NumbCoactivationMatrices; k ++)
	{
		for(int i = 0; i < *Host_NodeNumb; i++)
		{
			// initialize attribute for all nodes differs from start_node;
			double _Probability = (double)rand()/(double) RAND_MAX;
			if(_Probability < 0.333f)  Host_PrevNodeState[i + k*(*Host_NodeNumb)] = 1;
			else if( (0.333f <= _Probability ) && ( _Probability <= 0.666f ) ) Host_PrevNodeState[i + k*(*Host_NodeNumb)] = 0;
			else Host_PrevNodeState[i + k*(*Host_NodeNumb)] = -1;
		}
	}
	
	int NumberOfCoactivationEntries = NumbCoactivationMatrices*(*Host_NodeNumb)*((*Host_NodeNumb)-1)/2;
	float *Dev_CoactivationArray;
	HANDLE_ERROR(hipMalloc(&Dev_LinkListSize, (*Host_NodeNumb)*sizeof(int)));
	HANDLE_ERROR(hipMalloc(&Dev_CoactivationArray, NumberOfCoactivationEntries*sizeof(float)));
	int *Dev_PrevNodeState;
	HANDLE_ERROR(hipMalloc(&Dev_PrevNodeState, (*Host_NodeNumb)*NumbCoactivationMatrices*sizeof(int)));
	int *Dev_CurrNodeState;
	HANDLE_ERROR(hipMalloc(&Dev_CurrNodeState, (*Host_NodeNumb)*NumbCoactivationMatrices*sizeof(int)));
	// copy host data of previous state to device data
	HANDLE_ERROR(hipMemcpy(Dev_PrevNodeState, Host_PrevNodeState, (*Host_NodeNumb)*NumbCoactivationMatrices*sizeof(int), hipMemcpyHostToDevice));
	
	delete Host_NodeNumb;
	delete [] Host_LinkListSize;
	delete [] Host_LinkList;
	delete [] Host_FirstLocationNeighbor;

	/* done passing graph and allocate memory to device */
	
	/* Now run kernels */
	calCoactivationMat<<<128, 128>>>(Dev_MaxTimeStep, 
									Dev_NodeNumb, Dev_LinkListSize, 
									Dev_LinkList, Dev_FirstLocationNeighbor, 
									Dev_PrevNodeState, Dev_CurrNodeState, states, Dev_CoactivationArray);
	/* Done running kernel */
	
	/* Copy back to the host, write co-activation list to files */





	/* Done writing to files */
	// Now I can delete memory in GPU
	hipFree(Dev_MaxTimeStep);
	hipFree(Dev_NodeNumb);
	hipFree(Dev_LinkListSize);
	hipFree(Dev_PrevNodeState);
	hipFree(Dev_CurrNodeState);
	hipFree(Dev_CoactivationArray);

	std::cout << "Done forest fire simulation " << std::endl;
}

#endif